#include "hip/hip_runtime.h"


#include<algorithm>
#include <iostream>
#include <math.h>


using std::cout;
using std::endl;
#include "kmeanscu.cuh"




__device__ int random(unsigned int seed, int i)
{
	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
	 we will store a random state for every thread  */
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		i, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	
	return hiprand(&state) % DATASET_SIZE;
}


__global__ void randomCentroidsCuda(double cp_x[], double cp_y[], double* vect_x, double* vect_y, unsigned int seed)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= CLUSTER_SIZE) return;

	int rand;
	
	rand = random(seed, idx);

	// scelgo un punto randomico dal dataset e lo seleziono come centroide iniziale
	cp_x[idx] = vect_x[rand];
	cp_y[idx] = vect_y[rand];
}


__device__ double distance(double x1_point, double y1_point, double x2_point, double y2_point)
{
	return sqrt(pow(x1_point - x2_point, 2) + pow(y1_point - y2_point, 2));
}

__global__ void calculateDistanceCuda(double vect_x[], double vect_y[], double cp_x[], double cp_y[], int c_vect[])
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= DATASET_SIZE) return;

    double dist, temp;
    int cluster_class;

    __shared__ double s_vect_x[WRAPDIM];
    __shared__ double s_vect_y[WRAPDIM];

    __shared__ double s_vect_cx[CLUSTER_SIZE];
    __shared__ double s_vect_cy[CLUSTER_SIZE];

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < blockDim.x; i++)
        {
            if (i + idx >= DATASET_SIZE) break;

            s_vect_x[i] = vect_x[i+idx];
            s_vect_y[i] = vect_y[i+idx];
        }

        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            s_vect_cx[i] = cp_x[i];
            s_vect_cy[i] = cp_y[i];
        }
    }
    __syncthreads();

    
    // calculating distance between dataset point and centroid
    // selecting the centroid with minium distance

    dist = distance(s_vect_x[threadIdx.x], s_vect_y[threadIdx.x], s_vect_cx[0], s_vect_cy[0]);
    cluster_class = 0;
    
    for (int j = 0; j < CLUSTER_SIZE; j++)
    {
        temp = distance(s_vect_x[threadIdx.x], s_vect_y[threadIdx.x], s_vect_cx[j], s_vect_cy[j]);
        if (dist > temp) // looking for the minimum distance given a point
        {
            cluster_class = j;
            dist = temp;
        }
    }

    // updating to the beloging cluster 
    c_vect[idx] = cluster_class;
}

__global__ void updateCentroids(int vect_c[], double vect_x[], double vect_y[], double cp_x[], double cp_y[], int* change)
{
    double update_x, update_y;
    int num_points, count = 0;

    for (int i = 0; i < CLUSTER_SIZE; i++)
    {
        update_x = update_y = num_points = 0;

        for (int j = 0; j < DATASET_SIZE; j++)
        {
            if (vect_c[j] == i)
            {
                update_x += vect_x[j];
                update_y += vect_y[j];
                num_points++;
            }
        }

        // calculating che the center of the points given a cluster
        if (num_points != 0)
        {
            update_x = update_x / num_points;
            update_y = update_y / num_points;
        }

        //counting unchange centroid
        double cond = distance(cp_x[i], cp_y[i], update_x, update_y);


        if (cond <= THRESHOLD)
            count++;

        // updating centroids
        if (num_points != 0 && cond > THRESHOLD)
        {
            cp_x[i] = update_x;
            cp_y[i] = update_y;
        }

    }

    if (count > PERCENTAGE * CLUSTER_SIZE)
        *change = 1;
    else
        *change = 0;
}
