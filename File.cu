#include "hip/hip_runtime.h"


#include<algorithm>
#include <iostream>
#include <math.h>
#include "kmeanscu.cuh"

using std::cout;
using std::endl;






__device__ int random(unsigned int seed, int i)
{
	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
	 we will store a random state for every thread  */
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		i, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	
	return hiprand(&state) % DATASET_SIZE;
}


__global__ void randomCentroidsCuda(double cp_x[], double cp_y[], double* vect_x, double* vect_y, unsigned int seed)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= CLUSTER_SIZE) return;

	int rand;
	
	rand = random(seed, idx);

	// scelgo un punto randomico dal dataset e lo seleziono come centroide iniziale
	cp_x[idx] = vect_x[rand];
	cp_y[idx] = vect_y[rand];
}


__device__ double distance(double x1_point, double y1_point, double x2_point, double y2_point)
{
	return sqrt(pow(x1_point - x2_point, 2) + pow(y1_point - y2_point, 2));
}

__global__ void calculateDistanceCuda(double vect_x[], double vect_y[], double cp_x[], double cp_y[], int c_vect[])
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= DATASET_SIZE) return;

    double dist, temp;
    int cluster_class;

    __shared__ double s_vect_x[WRAPDIM];
    s_vect_x[threadIdx.x] = vect_x[idx];

    __shared__ double s_vect_y[WRAPDIM];
    s_vect_y[threadIdx.x] = vect_y[idx];

    __shared__ double s_vect_cx[CLUSTER_SIZE];
    __shared__ double s_vect_cy[CLUSTER_SIZE];

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            s_vect_cx[i] = cp_x[i];
            s_vect_cy[i] = cp_y[i];
        }
    }
    __syncthreads();

    // calculating distance between dataset point and centroid
    // selecting the centroid with minium distance

    dist = distance(s_vect_x[threadIdx.x], s_vect_y[threadIdx.x], s_vect_cx[0], s_vect_cy[0]);
    cluster_class = 0;
    
    for (int j = 0; j < CLUSTER_SIZE; j++)
    {
        temp = distance(s_vect_x[threadIdx.x], s_vect_y[threadIdx.x], s_vect_cx[j], s_vect_cy[j]);
        if (dist > temp) // looking for the minimum distance given a point
        {
            cluster_class = j;
            dist = temp;
        }
    }

    // updating to the beloging cluster 
    c_vect[idx] = cluster_class;

}

__global__ void updateCentroidsCuda(int vect_c[], double vect_x[], double vect_y[], double cp_x[], double cp_y[], int* change)
{
    double update_x, update_y;
    int num_points, count = 0;

    for (int i = 0; i < CLUSTER_SIZE; i++)
    {
        update_x = update_y = num_points = 0;

        for (int j = 0; j < DATASET_SIZE; j++)
        {
            if (vect_c[j] == i)
            {
                update_x += vect_x[j];
                update_y += vect_y[j];
                num_points++;
            }
        }

        // calculating che the center of the points given a cluster
        if (num_points != 0)
        {
            update_x = update_x / num_points;
            update_y = update_y / num_points;
        }

        //counting unchange centroid
        double cond = distance(cp_x[i], cp_y[i], update_x, update_y);


        if (cond <= THRESHOLD)
            count++;

        // updating centroids
        if (num_points != 0 && cond > THRESHOLD)
        {
            cp_x[i] = update_x;
            cp_y[i] = update_y;
        }

    }

    if (count > PERCENTAGE * CLUSTER_SIZE)
        *change = 1;
    else
        *change = 0;
}

__global__ void calculateCentroidMeans(int vect_c[], double vect_x[], double vect_y[], double sum_c_x[], double sum_c_y[], int num_c[])
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= DATASET_SIZE) return;

    __shared__ float s_vect_x[WRAPDIM];
    __shared__ float s_vect_y[WRAPDIM];
    __shared__ int s_vect_c[WRAPDIM];

   

    __shared__ double partial_sum_x[CLUSTER_SIZE];
    __shared__ double partial_sum_y[CLUSTER_SIZE];
    __shared__ int    partial_num[CLUSTER_SIZE];

    s_vect_x[threadIdx.x] = vect_x[idx];
    s_vect_c[threadIdx.x] = vect_c[idx];
    s_vect_y[threadIdx.x] = vect_y[idx];
    
   
    __syncthreads();

    if (threadIdx.x == 0)
    {
        int j;
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            partial_sum_x[i] = partial_sum_y[i] = partial_num[i] = 0;
        }
        
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            j = s_vect_c[i];
            if (j != -1)
            {
                partial_sum_x[j] += s_vect_x[i];
                partial_sum_y[j] += s_vect_y[i];
                partial_num[j] += 1;
            }
         
        }
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            atomicAdd(&sum_c_x[i], partial_sum_x[i]);
            
            atomicAdd(&sum_c_y[i], partial_sum_y[i]);

            atomicAdd(&num_c[i], partial_num[i]);
        }

    }


}



__global__ void updateC(double sum_c_x[], double sum_c_y[], int num_c[], double cp_x[], double cp_y[], double* count)
{
    
    // controllo della threashold
    // fare un atomic per incrementare il contatore di centroidi non modificati

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int c[WRAPDIM_C];

    if (idx >= CLUSTER_SIZE) return;

    // Calculating the means of the centroids
    sum_c_x[idx] = sum_c_x[idx] / num_c[idx];
    sum_c_y[idx] = sum_c_y[idx] / num_c[idx];

    // Checking the distance between the old and the new centroid

    double dist = distance(cp_x[idx], cp_y[idx], sum_c_x[idx], sum_c_y[idx]);

    __syncthreads();
    
    if (dist < THRESHOLD)
        c[threadIdx.x] = 1;
    else
    {
        c[threadIdx.x] = 0;
        cp_x[idx] = sum_c_x[idx];
        cp_y[idx] = sum_c_y[idx];
    }
    __syncthreads();  
    
    if (threadIdx.x == 0)
    {
        double sum = 0;
        for (int i = 0; i < WRAPDIM_C; i++)
        {
            sum += c[i];
        }
        

        atomicAdd(count,sum);
    }

}
