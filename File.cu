#include "hip/hip_runtime.h"


#include<algorithm>
#include <iostream>
#include <math.h>


using std::cout;
using std::endl;
#include "kmeanscu.cuh"


/*
__global__ void vectAdd(int* a, int* b, int* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
*/
/*__host__ int prova()
{
	int a[] = {1,2,3};
	int b[] = { 4,5,6 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;


	// alloccato memoria nel GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));


	// copiare i vettori nel gpu
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);


	vectAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	return 0;

}



__device__ double distance(double x1_point, double y1_point, double x2_point, double y2_point)
{
	return sqrt(pow(x1_point - x2_point, 2) + pow(y1_point - y2_point, 2));
}


__global__ void calculateDistanceCuda(double* vect_x, double* vect_y, centroid_point* cp, int* c_vect)
{
	double dist, temp;
	int cluster_class;

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx >= DATASET_SIZE) return;

	dist = distance(vect_x[idx], vect_y[idx], cp->x_c[0], cp->y_c[0]);
	cluster_class = 0;

	for (int j = 1; j < CLUSTER_SIZE; j++)
	{
		temp = distance(vect_x[idx], vect_y[idx], cp->x_c[j], cp->y_c[j]);

		if (dist > temp) // looking for the minimum distance given a point
		{
			cluster_class = j;
			dist = temp;
		}
	}

	// updating to the beloging cluster 
	c_vect[idx] = cluster_class;

	
}
*/

__device__ int random(unsigned int seed, int i)
{
	/* CUDA's random number library uses hiprandState_t to keep track of the seed value
	 we will store a random state for every thread  */
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		i, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	
	return hiprand(&state) % DATASET_SIZE;
}


__global__ void randomCentroidsCuda(double cp_x[], double cp_y[], double* vect_x, double* vect_y, unsigned int seed)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= CLUSTER_SIZE) return;

	int rand;
	
	rand = random(seed, idx);

	// scelgo un punto randomico dal dataset e lo seleziono come centroide iniziale
	cp_x[idx] = vect_x[rand];
	cp_y[idx] = vect_y[rand];
}


__device__ double distance(double x1_point, double y1_point, double x2_point, double y2_point)
{
	return sqrt(pow(x1_point - x2_point, 2) + pow(y1_point - y2_point, 2));
	//return (abs(x1_point-x2_point)+abs(y1_point-y2_point));
}

