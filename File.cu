#include "hip/hip_runtime.h"


#include "kmeanscu.cuh"


/*
__global__ void vectAdd(int* a, int* b, int* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
*/
/*__host__ int prova()
{
	int a[] = {1,2,3};
	int b[] = { 4,5,6 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;


	// alloccato memoria nel GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));


	// copiare i vettori nel gpu
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);


	vectAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	for (int i = 0; i < 3; i++)
	{
		std::cout << c[i] << std::endl;
	}
	

	

	return 0;

}



__device__ double distance(double x1_point, double y1_point, double x2_point, double y2_point)
{
	return sqrt(pow(x1_point - x2_point, 2) + pow(y1_point - y2_point, 2));
}


__global__ void calculateDistanceCuda(double* vect_x, double* vect_y, centroid_point* cp, int* c_vect)
{
	double dist, temp;
	int cluster_class;

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx >= DATASET_SIZE) return;

	dist = distance(vect_x[idx], vect_y[idx], cp->x_c[0], cp->y_c[0]);
	cluster_class = 0;

	for (int j = 1; j < CLUSTER_SIZE; j++)
	{
		temp = distance(vect_x[idx], vect_y[idx], cp->x_c[j], cp->y_c[j]);

		if (dist > temp) // looking for the minimum distance given a point
		{
			cluster_class = j;
			dist = temp;
		}
	}

	// updating to the beloging cluster 
	c_vect[idx] = cluster_class;

	
}*/