#include "hip/hip_runtime.h"


#include<algorithm>
#include <iostream>
#include <math.h>
#include "kmeanscu.cuh"

using std::cout;
using std::endl;






__device__ double distance(double x1_point, double y1_point, double x2_point, double y2_point)
{
	return sqrt(pow(x1_point - x2_point, 2) + pow(y1_point - y2_point, 2));
}

__global__ void calculateDistanceCuda(double vect_x[], double vect_y[], double cp_x[], double cp_y[], int c_vect[])
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= DATASET_SIZE) return;

    double dist, temp;
    int cluster_class;

    __shared__ double s_vect_cx[CLUSTER_SIZE];
    __shared__ double s_vect_cy[CLUSTER_SIZE];

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            s_vect_cx[i] = cp_x[i];
            s_vect_cy[i] = cp_y[i];
        }
    }
    __syncthreads();

    // calculating distance between dataset point and centroid
    // selecting the centroid with minium distance

    dist = distance(vect_x[idx], vect_y[idx], s_vect_cx[0], s_vect_cy[0]);
    cluster_class = 0;
    
    for (int j = 0; j < CLUSTER_SIZE; j++)
    {
        temp = distance(vect_x[idx], vect_y[idx], s_vect_cx[j], s_vect_cy[j]);
        // looking for the minimum distance given a point
        if (dist > temp) 
        {
            cluster_class = j;
            dist = temp;
        }
    }
    // updating to the beloging cluster 
    c_vect[idx] = cluster_class;

}


__global__ void calculateCentroidMeans(int vect_c[], double vect_x[], double vect_y[], double sum_c_x[], double sum_c_y[], int num_c[])
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= DATASET_SIZE) return;

    __shared__ double s_vect_x[BLOCK];
    __shared__ double s_vect_y[BLOCK];
    __shared__ int s_vect_c[BLOCK];

    __shared__ double partial_sum_x[CLUSTER_SIZE];
    __shared__ double partial_sum_y[CLUSTER_SIZE];
    __shared__ int    partial_num[CLUSTER_SIZE];

    // metto nella shared memory la porzione di dati che il blocco usera per fare i calcoli
    s_vect_x[threadIdx.x] = vect_x[idx];
    s_vect_c[threadIdx.x] = vect_c[idx];
    s_vect_y[threadIdx.x] = vect_y[idx];
    
    __syncthreads();

    if (threadIdx.x == 0)
    {
        int j;
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            partial_sum_x[i] = 0;
            partial_sum_y[i] = 0;
            partial_num[i] = 0;
        }
        
  
       
        int q = 0;
        if (DATASET_SIZE - (blockIdx.x * BLOCK) < BLOCK)
            q = DATASET_SIZE - (blockIdx.x * BLOCK);
        else
            q = BLOCK;

        for (int i = 0; i < q; i++)
        {
            j = s_vect_c[i];
            partial_sum_x[j] += s_vect_x[i];
            partial_sum_y[j] += s_vect_y[i];
            partial_num[j] += 1;
        }
        for (int i = 0; i < CLUSTER_SIZE; i++)
        {
            atomicAdd(&sum_c_x[i], partial_sum_x[i]);
            atomicAdd(&sum_c_y[i], partial_sum_y[i]);
            atomicAdd(&num_c[i], partial_num[i]);
        }

    }
}



__global__ void updateC(double sum_c_x[], double sum_c_y[], int num_c[], double cp_x[], double cp_y[], double* count)
{

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int c[BLOCK_C];

    if (idx >= CLUSTER_SIZE){return;}

    for (int i = 0; i < BLOCK_C; i++){c[i] = 0;}

    // Calculating the means of the centroids
    if (num_c[idx] == 0){num_c[idx] = 1;}
    
    sum_c_x[idx] = sum_c_x[idx] / num_c[idx];
    sum_c_y[idx] = sum_c_y[idx] / num_c[idx];
    
    // Checking the distance between the old and the new centroid

    double dist = distance(cp_x[idx], cp_y[idx], sum_c_x[idx], sum_c_y[idx]);
    
    if (dist <= THRESHOLD){c[threadIdx.x] = 1;}
    else
    {
        c[threadIdx.x] = 0;
        cp_x[idx] = sum_c_x[idx];
        cp_y[idx] = sum_c_y[idx];
    }
    
    __syncthreads();
    // caluculating unchange centroids
    if (threadIdx.x == 0)
    {
        double sum = 0;
        for (int i = 0; i < BLOCK_C; i++)
        { sum += c[i]; } 
        atomicAdd(count,sum);
    }
    // setting the partial sum vectors to 0
    sum_c_x[idx] = 0;
    sum_c_y[idx] = 0;
    num_c[idx] = 0;
}


__global__ void updateS(double sum_c_x[], double sum_c_y[], int num_c[], double cp_x[], double cp_y[], double* count)
{

    int c = 0;
    double dist = 0;
    double tmp_x = 0;
    double tmp_y = 0;

    for (int i = 0; i < CLUSTER_SIZE; i++)
    {
        if (num_c[i] != 0){

            tmp_x = sum_c_x[i] / num_c[i];
            tmp_y = sum_c_y[i] / num_c[i];
            dist = distance(tmp_x, tmp_y, cp_x[i], cp_y[i]);

            if (dist <= THRESHOLD) { c++; }
            else
            {
                cp_x[i] = tmp_x;
                cp_y[i] = tmp_y;
            }
            sum_c_x[i] = 0;
            sum_c_y[i] = 0;
            num_c[i] = 0;
        }
    }
    *count = c;
}