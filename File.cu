#include<hip/hip_runtime.h>

#include <iostream>

__global__ void vectAdd(int* a, int* b, int* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	int a[] = { 1,2,3 };
	int b[] = { 4,5,6 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;


	// alloccato memoria nel GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));


	// copiare i vettori nel gpu
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);


	vectAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	for (int i = 0; i < 3; i++)
	{
		std::cout << c[i] << std::endl;
	}
	return 0;

}