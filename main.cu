


#include"Intestazione.h"

#include"kmeanscu.cuh"




int main()
{
    std::string   DATASET_PATH;

    // if the centroid are not changed then the method stops
    bool isChange = true;
    int i = 0;
    
    cout << "DATASET SIZE: " << DATASET_SIZE << " CLUSTER SIZE: " << CLUSTER_SIZE << " ITERATIONS: " << endl;

    // declaring array for dataset point and for centroid points
    
    double  x[DATASET_SIZE];
    double  y[DATASET_SIZE];
    int     c[DATASET_SIZE];
    
    double cpx[CLUSTER_SIZE];
    double cpy[CLUSTER_SIZE];


    // -----------------------------------------
    //alloccare memoria nella GPU

    double* cudax = 0;
    double* cuday = 0;
    int*    cudac = 0;

    double* cudacpx = 0;
    double* cudacpy = 0;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&cudax, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cuday, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cudac, DATASET_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&cudacpx, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cudacpy, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // -----------------------------------------
    
   
    DATASET_PATH = "Datasets/dataset/ds.txt";
    loadDataset(DATASET_PATH, x, y, c);

    cout << "Finish loading data.." << endl;

    // copia del dataset nella gpu
  
    cudaStatus = hipMemcpy(cudax, x, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cuday, y, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cudac, c, DATASET_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // -----------------------------------------------------


    // generating random centroid for the first step of the method
    cout << "Generating first " << CLUSTER_SIZE << " centroids.." << endl;


    // chiamare il randomcentroidcuda
    
    //randomCentroids(cpx, cpy, x, y);

    randomCentroidsCuda <<< (CLUSTER_SIZE+32)/32, 32 >> > (cudacpx, cudacpy, cudax, cuday, time(NULL));


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

  
    // -----------------------------------------
    // copiare i centroidi iniziali nella gpu
    //hipMemcpy(cudacpx, cpx, sizeof(cpx), hipMemcpyHostToDevice);
    //hipMemcpy(cudacpy, cpy, sizeof(cpy), hipMemcpyHostToDevice);

    // -----------------------------------------

       // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    cout << "Finish generating random centroids.." << endl;

    printClusterPoint(cpx, cpy);


    



    while (isChange == true)
    {
        cout << "Calculating cluster cycle: " << i + 1 << "..." << endl;
        calculateDistance(x, y, cpx, cpy, c);
        cout << "End calculating cluster cycle: " << i + 1 << endl;

        cout << "Updating centroids..." << endl;
        isChange = updateCentroids(c, x, y, cpx, cpy);
        cout << "End Updating centroids..." << endl;
        i++;

    }



    // copio i cluster dei punti e i centroidi dalla gpu alla memoria ram
    //hipMemcpy(c, cudac, sizeof(cudac), hipMemcpyDeviceToHost);
    //hipMemcpy(cpx, cudacpx, sizeof(cudacpx), hipMemcpyDeviceToHost);
    //hipMemcpy(cpy, cudacpy, sizeof(cudacpy), hipMemcpyDeviceToHost);


    // printing the centroid after the kmeans methods
    printClusterPoint(cpx, cpy);


Error:
    // free dei puntatori
    hipFree(cudax);
    hipFree(cuday);
    hipFree(cudac);
    hipFree(cudacpx);
    hipFree(cudacpy);





}