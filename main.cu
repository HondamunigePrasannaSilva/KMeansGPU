


#include"Intestazione.h"

#include"kmeanscu.cuh"




int main()
{
    std::string   DATASET_PATH;

    // if the centroid are not changed then the method stops
    int* isChange = (int*)malloc(sizeof(int));
    
    *isChange = 0;

    int i = 0;
    
    cout << "DATASET SIZE: " << DATASET_SIZE << " CLUSTER SIZE: " << CLUSTER_SIZE << " ITERATIONS: " << endl;

    // declaring array for dataset point and for centroid points
    
    double  x[DATASET_SIZE];
    double  y[DATASET_SIZE];
    int     c[DATASET_SIZE];
    
    double cpx[CLUSTER_SIZE];
    double cpy[CLUSTER_SIZE];


    // -----------------------------------------
    //alloccare memoria nella GPU

    double* cudax = 0;
    double* cuday = 0;
    int*    cudac = 0;

    double* cudacpx = 0;
    double* cudacpy = 0;

    int*    change = 0;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&cudax, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc vect_x failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cuday, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc vect_y failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cudac, DATASET_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc vect_c  failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&cudacpx, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc vect_cpx failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&cudacpy, DATASET_SIZE * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc vect_cpx failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&change, DATASET_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc vect_cpx failed!");
        goto Error;
    }



    // -----------------------------------------
    
   
    DATASET_PATH = "Datasets/dataset/ds.txt";
    loadDataset(DATASET_PATH, x, y, c);

    cout << "Finish loading data.." << endl;

    // copia del dataset nella gpu
  
    cudaStatus = hipMemcpy(cudax, x, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_x failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cuday, y, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_y failed!");
        goto Error;
    }
   
    // -----------------------------------------------------


    // generating random centroid for the first step of the method
    cout << "Generating first " << CLUSTER_SIZE << " centroids.." << endl;


    
    randomCentroidsCuda <<< (CLUSTER_SIZE+32)/32, 32 >> > (cudacpx, cudacpy, cudax, cuday, time(NULL));


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "randomCentroidCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching randomCentroidCuda!\n", cudaStatus);
        goto Error;
    }

  
    

    // -----------------------------------------

       // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpx failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpy failed!");
        goto Error;
    }
    
    cout << "Finish generating random centroids.." << endl;

    printClusterPoint(cpx, cpy);


    



    while (*isChange == 0)
    {
        cout << "Calculating cluster cycle: " << i + 1 << "..." << endl;
        cout << BLOCKDIM;
        calculateDistanceCuda<<<BLOCKDIM, WRAPDIM >>>(cudax, cuday, cudacpx, cudacpy, cudac);


        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateDistanceCuda!\n", cudaStatus);
            goto Error;
        }


        cout << "End calculating cluster cycle: " << i + 1 << endl;

        cout << "Updating centroids..." << endl;

        //isChange = updateCentroids(c, x, y, cpx, cpy);
        updateCentroids<<<1,1>>>(cudac, cudax, cuday, cudacpx, cudacpy, change);
        
        cout << "End Updating centroids..." << endl;
        i++;

        cudaStatus = hipMemcpy(isChange, change, sizeof(int), hipMemcpyDeviceToHost);
       
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy change failed!");
            goto Error;
        }

    }



    // copio i cluster dei punti e i centroidi dalla gpu alla memoria ram
    //hipMemcpy(c, cudac, sizeof(cudac), hipMemcpyDeviceToHost);
    //hipMemcpy(cpx, cudacpx, sizeof(cudacpx), hipMemcpyDeviceToHost);
    //hipMemcpy(cpy, cudacpy, sizeof(cudacpy), hipMemcpyDeviceToHost);

    
    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpx failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(c, cudac, DATASET_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_c failed!");
        goto Error;
    }
    



    // printing the centroid after the kmeans methods
    printClusterPoint(cpx, cpy);


Error:
    // free dei puntatori
    hipFree(cudax);
    hipFree(cuday);
    hipFree(cudac);
    hipFree(cudacpx);
    hipFree(cudacpy);





}