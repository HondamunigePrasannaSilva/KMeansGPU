


#include"Intestazione.h"

#include"kmeanscu.cuh"



int main()
{
    std::string   DATASET_PATH;
    DATASET_PATH = "Datasets/dataset2/ds.txt";

    double* count = (double*)malloc(sizeof(double));

    *count = 0;

    int i = 0;
    
    cout << "DATASET SIZE: " << DATASET_SIZE << " CLUSTER SIZE: " << CLUSTER_SIZE << " ITERATIONS: " << endl;

    // declaring array for dataset point and for centroid points
    
    static double  x[DATASET_SIZE];
    static double  y[DATASET_SIZE];
    static int     c[DATASET_SIZE];
    
    double cpx[CLUSTER_SIZE];   // array of centroid, x value
    double cpy[CLUSTER_SIZE];   // array of centroid, y value

  
    // -----------------------------------------
    //alloccare memoria nella GPU

    double* cudax = 0;
    double* cuday = 0;
    int*    cudac = 0;

    double* cudacpx = 0;
    double* cudacpy = 0;

    double* cudascx = 0;
    double* cudascy = 0;
    int*    cudanc  = 0;

    int*    change = 0;

    double*    cudacount = 0;

    hipError_t cudaStatus;
    
    cudaStatus = hipMalloc((void**)&cudax, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudax")) goto Error;
   

    cudaStatus = hipMalloc((void**)&cuday, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cuday")) goto Error;

    cudaStatus = hipMalloc((void**)&cudac, DATASET_SIZE * sizeof(int));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudac")) goto Error;
   
    cudaStatus = hipMalloc((void**)&cudacpx, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudacpx")) goto Error;
   

    cudaStatus = hipMalloc((void**)&cudacpy, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudacpy")) goto Error;


    cudaStatus = hipMalloc((void**)&change, sizeof(int));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "change")) goto Error;

    cudaStatus = hipMalloc((void**)&cudacount, sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "count")) goto Error;


    cudaStatus = hipMalloc((void**)&cudascx, CLUSTER_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudascx")) goto Error;

    cudaStatus = hipMalloc((void**)&cudascy, CLUSTER_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudascy")) goto Error;

    cudaStatus = hipMalloc((void**)&cudanc, CLUSTER_SIZE * sizeof(int));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudanc")) goto Error;


    // initialize sum of centroid and number of data of each centroid to 0.
    hipMemset(cudascx, 0, CLUSTER_SIZE * sizeof(double));
    hipMemset(cudascy, 0, CLUSTER_SIZE * sizeof(double));
    hipMemset(cudanc, 0,  CLUSTER_SIZE * sizeof(int));

    hipMemset(cudacount, 0,  sizeof(double));
    hipMemset(cudac, -1, sizeof(int));


    


    // -----------------------------------------
    
   
    
    loadDataset(DATASET_PATH, x, y, c);

    cout << "Finish loading data.." << endl;

    // copia del dataset nella gpu
  
    cudaStatus = hipMemcpy(cudax, x, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaErrorStatus("hipMemcpy", cudaStatus, "x->cudax")) goto Error;

    cudaStatus = hipMemcpy(cuday, y, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaErrorStatus("hipMemcpy", cudaStatus, "y->cuday")) goto Error;

   
    // -----------------------------------------------------


    // generating random centroid for the first step of the method
    cout << "Generating first " << CLUSTER_SIZE << " centroids.." << endl;

    randomCentroidsCuda <<< BLOCKDIM_C, WRAPDIM_C >> > (cudacpx, cudacpy, cudax, cuday, time(NULL));

    cudaStatus = hipGetLastError();
    if (cudaErrorStatus("randomCentroidCuda", cudaStatus, hipGetErrorString(cudaStatus))) goto Error;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching randomCentroidCuda!\n", cudaStatus);
        goto Error;
    }
  
    

    // -----------------------------------------

       // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpx failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpy failed!");
        goto Error;
    }
    
    cout << "Finish generating random centroids.." << endl;

    printClusterPoint(cpx, cpy);


    



    while ((int) *count < PERCENTAGE * CLUSTER_SIZE)
    //while (*isChange == 0)
    {
        
        cout << "Calculating cluster cycle: " << i + 1 << "..." << endl;
      
        calculateDistanceCuda<<<BLOCKDIM, WRAPDIM >>>(cudax, cuday, cudacpx, cudacpy, cudac);


        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateDistanceCuda!\n", cudaStatus);
            goto Error;
        }

        cout << "End calculating cluster cycle: " << i + 1 << endl;

        cout << "Updating centroids..." << endl;

        
        calculateCentroidMeans << <BLOCKDIM, WRAPDIM >> >(cudac, cudax, cuday, cudascx, cudascy, cudanc);

        cudaStatus = hipGetLastError();
        if (cudaErrorStatus("calculateCentroidMeans ", cudaStatus, hipGetErrorString(cudaStatus))) goto Error;


        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateCentroidMeans!\n", cudaStatus);
            goto Error;
        }

        hipMemset(cudacount, 0, sizeof(double));
        *count = 0;
        cout << "count I " << (int) *count << endl;
        
        updateC<<<BLOCKDIM_C, WRAPDIM_C>>>(cudascx, cudascy, cudanc,cudacpx, cudacpy, cudacount);


        hipMemset(cudascx, 0, CLUSTER_SIZE * sizeof(double));
        hipMemset(cudascy, 0, CLUSTER_SIZE * sizeof(double));
        hipMemset(cudanc, 0, CLUSTER_SIZE * sizeof(int));


        cudaStatus = hipGetLastError();
        if (cudaErrorStatus("updateC ", cudaStatus, hipGetErrorString(cudaStatus))) goto Error;


        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updateC!\n", cudaStatus);
            goto Error;
        }

      
       
        cudaStatus = hipMemcpy(count, cudacount, sizeof(double), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy count failed!");
            goto Error;
        }
        cout << "count F " <<*count << endl;

        //updateCentroidsCuda <<<1,1>>>(cudac, cudax, cuday, cudacpx, cudacpy, change);
        //cudaStatus = hipMemcpy(isChange, change, sizeof(int), hipMemcpyDeviceToHost);


        cout << "End Updating centroids..." << endl;
        i++;

    

    }

    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpx failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(c, cudac, DATASET_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_c failed!");
        goto Error;
    }
    



    // printing the centroid after the kmeans methods
    printClusterPoint(cpx, cpy);
    

    free(count);

Error:
    // free dei puntatori
    hipFree(cudax);
    hipFree(cuday);
    hipFree(cudac);
    hipFree(cudacpx);
    hipFree(cudacpy);
    hipFree(cudascx);
    hipFree(cudascy);
    hipFree(cudanc);
    hipFree(change);
    hipFree(cudacount);





}