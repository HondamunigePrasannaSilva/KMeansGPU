


#include"Intestazione.h"

#include"kmeanscu.cuh"




int main()
{
    std::string   DATASET_PATH;

    // if the centroid are not changed then the method stops
    int* isChange = (int*)malloc(sizeof(int));
    
    *isChange = 0;

    int i = 0;
    
    cout << "DATASET SIZE: " << DATASET_SIZE << " CLUSTER SIZE: " << CLUSTER_SIZE << " ITERATIONS: " << endl;

    // declaring array for dataset point and for centroid points
    
    double  x[DATASET_SIZE];
    double  y[DATASET_SIZE];
    int     c[DATASET_SIZE];
    
    double cpx[CLUSTER_SIZE];   // array of centroid, x value
    double cpy[CLUSTER_SIZE];   // array of centroid, y value

    double sum_cen_x[CLUSTER_SIZE];
    double sum_cen_y[CLUSTER_SIZE];
    int    num_cen[CLUSTER_SIZE];


    // -----------------------------------------
    //alloccare memoria nella GPU

    double* cudax = 0;
    double* cuday = 0;
    int*    cudac = 0;

    double* cudacpx = 0;
    double* cudacpy = 0;

    double* cudascx = 0;
    double* cudascy = 0;
    int* cudanc  = 0;

    int*    change = 0;
    hipError_t cudaStatus;
    
    cudaStatus = hipMalloc((void**)&cudax, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudax")) goto Error;
   

    cudaStatus = hipMalloc((void**)&cuday, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cuday")) goto Error;

    cudaStatus = hipMalloc((void**)&cudac, DATASET_SIZE * sizeof(int));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudac")) goto Error;
   
    cudaStatus = hipMalloc((void**)&cudacpx, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudacpx")) goto Error;
   

    cudaStatus = hipMalloc((void**)&cudacpy, DATASET_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudacpy")) goto Error;


    cudaStatus = hipMalloc((void**)&change, sizeof(int));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "change")) goto Error;



    cudaStatus = hipMalloc((void**)&cudascx, CLUSTER_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudascx")) goto Error;

    cudaStatus = hipMalloc((void**)&cudascy, CLUSTER_SIZE * sizeof(double));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudascy")) goto Error;

    cudaStatus = hipMalloc((void**)&cudanc, CLUSTER_SIZE * sizeof(int));
    if (cudaErrorStatus("hipMalloc", cudaStatus, "cudanc")) goto Error;


    // initialize sum of centroid and number of data of each centroid to 0.
    hipMemset(cudascx, 0, CLUSTER_SIZE * sizeof(double));
    hipMemset(cudascy, 0, CLUSTER_SIZE * sizeof(double));
    hipMemset(cudanc, 0,  CLUSTER_SIZE * sizeof(int));


    // -----------------------------------------
    
   
    DATASET_PATH = "Datasets/dataset/ds.txt";
    loadDataset(DATASET_PATH, x, y, c);

    cout << "Finish loading data.." << endl;

    // copia del dataset nella gpu
  
    cudaStatus = hipMemcpy(cudax, x, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaErrorStatus("hipMemcpy", cudaStatus, "x->cudax")) goto Error;

    cudaStatus = hipMemcpy(cuday, y, DATASET_SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (cudaErrorStatus("hipMemcpy", cudaStatus, "y->cuday")) goto Error;

   
    // -----------------------------------------------------


    // generating random centroid for the first step of the method
    cout << "Generating first " << CLUSTER_SIZE << " centroids.." << endl;


    
    randomCentroidsCuda <<< (CLUSTER_SIZE+32)/32, 32 >> > (cudacpx, cudacpy, cudax, cuday, time(NULL));


    cudaStatus = hipGetLastError();
    if (cudaErrorStatus("randomCentroidCuda", cudaStatus, hipGetErrorString(cudaStatus))) goto Error;

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching randomCentroidCuda!\n", cudaStatus);
        goto Error;
    }
  
    

    // -----------------------------------------

       // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpx failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpy failed!");
        goto Error;
    }
    
    cout << "Finish generating random centroids.." << endl;

    printClusterPoint(cpx, cpy);


    



    while (*isChange == 0)
    {
        cout << "Calculating cluster cycle: " << i + 1 << "..." << endl;
        cout << BLOCKDIM;
        calculateDistanceCuda<<<BLOCKDIM, WRAPDIM >>>(cudax, cuday, cudacpx, cudacpy, cudac);


        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateDistanceCuda!\n", cudaStatus);
            goto Error;
        }


        cout << "End calculating cluster cycle: " << i + 1 << endl;

        cout << "Updating centroids..." << endl;

        //updateCentroids<<<1,1>>>(cudac, cudax, cuday, cudacpx, cudacpy, change);
        
        calculateCentroidMeans<<<BLOCKDIM , WRAPDIM>>>(cudac, cudax, cuday, cudascx, cudascy, cudanc);
        cudaStatus = hipGetLastError();
        if (cudaErrorStatus("calculateCentroidMeans ", cudaStatus, hipGetErrorString(cudaStatus))) goto Error;


        

        



        cout << "End Updating centroids..." << endl;
        i++;

        /*
        cudaStatus = hipMemcpy(isChange, change, sizeof(int), hipMemcpyDeviceToHost);
       
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy change failed!");
            goto Error;
        }*/

    }

    cudaStatus = hipMemcpy(cpx, cudacpx, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpx failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(cpy, cudacpy, CLUSTER_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_cpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(c, cudac, DATASET_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy vect_c failed!");
        goto Error;
    }
    



    // printing the centroid after the kmeans methods
    printClusterPoint(cpx, cpy);


Error:
    // free dei puntatori
    hipFree(cudax);
    hipFree(cuday);
    hipFree(cudac);
    hipFree(cudacpx);
    hipFree(cudacpy);





}